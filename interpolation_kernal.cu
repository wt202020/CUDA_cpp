#include "hip/hip_runtime.h"
#include <torch/extension.h>


template <typename scalar_t>
__global__ void trilinear_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> point,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp
)
{
    const int N = feats.size(0);
    const int F = feats.size(2);

    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;

    if (n < N && f < F)
    {
        const scalar_t u = (point[n][0] + 1) / 2;
        const scalar_t v = (point[n][1] + 1) / 2;
        const scalar_t w = (point[n][2] + 1) / 2;

        const scalar_t a = (1-v) * (1-w);
        const scalar_t b = (1-v) * w;
        const scalar_t c = v * (1-w);
        const scalar_t d = 1 - a - b - c;

        feat_interp[n][f] = (1-u) * (a * feats[n][0][f] + b * feats[n][1][f] + c * feats[n][2][f] + d * feats[n][3][f]) + 
                            u * (a * feats[n][4][f] + b * feats[n][5][f] + c * feats[n][6][f] + d * feats[n][7][f]);
    }
}

torch::Tensor trilinear_fw_cu(
    torch::Tensor feats,
    torch::Tensor point
)
{ 
    const int N = feats.size(0);
    const int F = feats.size(2);

    torch::Tensor feat_interp = torch::zeros({N, F}, feats.options());

    const dim3 threads(16, 16);
    const dim3 blocks((N + threads.x - 1) / threads.x, (F + threads.y - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_fw_cu", 
    ([&] {
        trilinear_fw_kernel<scalar_t><<<blocks, threads>>>(      //scalar_t represent the type of the input tensor that maybe unkonwn
            feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(), 
            point.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            feat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return feat_interp;
}